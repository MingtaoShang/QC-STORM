#include "DH3D_MoleculePair.h"


void DH3D_MoleculePair_TypeDef::MoleculePair(float *h_iLocArry, int FluoNum, LocalizationPara & LocPara, hipStream_t cstream)
{
	hipMemsetAsync(d_ValidoFluoNum, 0, sizeof(int), cstream);

	hipMemcpyAsync(d_LocArry, h_iLocArry, FluoNum*OutParaNumGS2D * sizeof(float), hipMemcpyHostToDevice, cstream);


	// pair molecules
	int BlockDim = ThreadsPerBlock;
	int BlockNum = ((FluoNum + ThreadsPerBlock - 1) / ThreadsPerBlock);

	gpu_MoleculePair << <BlockNum, BlockDim, 0, cstream >> >(d_LocArry, FluoNum, d_PairID, d_ValidoFluoNum, LocPara.DH_MeanDistance, LocPara.DH_DistanceTh);

	hipMemcpyAsync(h_ValidoFluoNum, d_ValidoFluoNum, sizeof(int), hipMemcpyDeviceToHost, cstream);
	hipStreamSynchronize(cstream);

	oValidFluoNum = *h_ValidoFluoNum;

	// convert paired molecules into single localization


	BlockDim = ThreadsPerBlock;
	BlockNum = ((oValidFluoNum + ThreadsPerBlock - 1) / ThreadsPerBlock);

	gpu_MoleculeMerge << <BlockNum, BlockDim, 0, cstream >> > (d_LocArry, d_oLocArry, d_PairID, oValidFluoNum, LocPara.DH_RotateType, LocPara.p4_XGY, LocPara.p3_XGY, LocPara.p2_XGY, LocPara.p1_XGY, LocPara.p0_XGY);

	hipMemcpyAsync(h_oLocArry, d_oLocArry, oValidFluoNum*OutParaNumGS2D * sizeof(float), hipMemcpyDeviceToHost, cstream);

	hipStreamSynchronize(cstream);

}


void DH3D_MoleculePair_TypeDef::Init()
{
	hipHostMalloc((void **)&h_LocArry, MaxPointNum * OutParaNumGS2D * sizeof(float));
	hipMalloc((void **)&d_LocArry, MaxPointNum * OutParaNumGS2D * sizeof(float));

	hipHostMalloc((void **)&h_oLocArry, (MaxPointNum / 2) * OutParaNumGS2D * sizeof(float));
	hipMalloc((void **)&d_oLocArry, (MaxPointNum / 2) * OutParaNumGS2D * sizeof(float));


	hipHostMalloc((void **)&h_PairID, (MaxPointNum / 2) * PAIR_ID_LEN * sizeof(int));
	hipMalloc((void **)&d_PairID, (MaxPointNum / 2)  *PAIR_ID_LEN * sizeof(int));

	hipHostMalloc((void **)&h_ValidoFluoNum, sizeof(int));
	hipMalloc((void **)&d_ValidoFluoNum, sizeof(int));

	oValidFluoNum = 0;

}

void DH3D_MoleculePair_TypeDef::Deinit()
{
	hipError_t err;

	err = hipHostFree(h_LocArry);
	err = hipFree(d_LocArry);

	err = hipHostFree(h_oLocArry);
	err = hipFree(d_oLocArry);


	err = hipHostFree(h_PairID);
	err = hipFree(d_PairID);

	err = hipHostFree(h_ValidoFluoNum);
	err = hipFree(d_ValidoFluoNum);

}

