/*
This program is free software: you can redistribute it and/or modify
it under the terms of the GNU LESSER GENERAL PUBLIC LICENSE as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU LESSER GENERAL PUBLIC LICENSE for more details.

You should have received a copy of the GNU LESSER GENERAL PUBLIC LICENSE
along with this program.  If not, see <https://www.gnu.org/licenses/>.
*/

#include "ConsecutiveFit.h"
#include "bfgs_top.h"



// d_iLocArry come from localization data 
void ConsecutiveFit_TypeDef::ConsecutiveFit_WeightedAvg(float * h_iLocArry, int FluoNum_CurGrop, int IsEndProc, LocalizationPara & LocPara, hipStream_t cstream)
{
	int TotalFluoNum = FluoNum_LastGroup + FluoNum_CurGrop;

	// copy data, find consecutive molecules should process the acjecent molecules between two independent image groups
	hipMemcpyAsync(d_LocArry_ConsecFit, d_LocArry_LastGroup, FluoNum_LastGroup*OutParaNumGS2D*sizeof(float), hipMemcpyDeviceToDevice, cstream);

	hipMemcpyAsync(&d_LocArry_ConsecFit[FluoNum_LastGroup*OutParaNumGS2D], h_iLocArry, FluoNum_CurGrop*OutParaNumGS2D * sizeof(float), hipMemcpyHostToDevice, cstream);


	// find consecutive molecules
	float DistanceTh_Pixel = LocPara.ConsecFit_DistanceTh_nm / LocPara.PixelSize;

	int BlockDim = ThreadsPerBlock;
	int BlockNum = ((TotalFluoNum + ThreadsPerBlock - 1) / ThreadsPerBlock);
	
	hipMemsetAsync(d_ForwardLinkID, 0, TotalFluoNum * sizeof(int), cstream);
	hipMemsetAsync(d_BackwardLinkID, 0, TotalFluoNum * sizeof(int), cstream);

	// ontime calculation
	hipMemsetAsync(d_OntimeDistrib, 0, MaxOnTimeConsecutiveNum * sizeof(int), cstream);
	hipMemsetAsync(d_ValidFluoNum, 0, sizeof(int), cstream);


	gpuFindConsecutiveMolecules << <BlockNum, BlockDim, 0, cstream >> > (d_LocArry_ConsecFit, TotalFluoNum, d_ForwardLinkID, d_BackwardLinkID, DistanceTh_Pixel);
	
	gpuConsecutiveFit << <BlockNum, BlockDim, 0, cstream >> > (d_LocArry_ConsecFit, TotalFluoNum, FluoNum_LastGroup, d_ForwardLinkID, d_BackwardLinkID, d_OntimeDistrib, d_ValidFluoNum, LocPara.QE);


	gpuRemoveConsecutiveFluo << <BlockNum, BlockDim, 0, cstream >> > (d_LocArry_ConsecFit, TotalFluoNum, d_ForwardLinkID, d_BackwardLinkID);

	hipMemcpyAsync(h_ValidFluoNum, d_ValidFluoNum, sizeof(int), hipMemcpyDeviceToHost, cstream);
	hipMemcpyAsync(h_OntimeDistrib, d_OntimeDistrib, MaxOnTimeConsecutiveNum * sizeof(int), hipMemcpyDeviceToHost, cstream);


	hipStreamSynchronize(cstream);

	// ontime distribution ratio
	if (*h_ValidFluoNum <= 0)*h_ValidFluoNum = 1;
	for (int cnt = 0; cnt < MaxOnTimeConsecutiveNum; cnt++)
	{
		h_OntimeRatio[cnt] = h_OntimeDistrib[cnt] * 1.0f / (*(h_ValidFluoNum));
	}


	// localization precision update for consecutive fitted molecules
	LDLocData_TypeDef::LocPrecCalc_GaussianCRLB(d_LocArry_ConsecFit, LocPara, FluoNum_LastGroup, cstream);


	if (IsEndProc == 0)
	{
		// output last group
		OutFluoNum = FluoNum_LastGroup;

		hipMemcpyAsync(d_LocArry_LastGroup, &d_LocArry_ConsecFit[FluoNum_LastGroup*OutParaNumGS2D], FluoNum_CurGrop*OutParaNumGS2D * sizeof(float), hipMemcpyDeviceToDevice, cstream);
		
		FluoNum_LastGroup = FluoNum_CurGrop;
	}
	else
	{
		// output all
		OutFluoNum = TotalFluoNum;
		FluoNum_LastGroup = 0;
	}

	hipMemcpyAsync(h_OutLocArry, d_LocArry_ConsecFit, OutFluoNum*OutParaNumGS2D * sizeof(float), hipMemcpyDeviceToHost, cstream);
	hipStreamSynchronize(cstream);

}



void ConsecutiveFit_TypeDef::Init()
{
	OutFluoNum = 0;
	FluoNum_LastGroup = 0;

	hipHostMalloc((void **)&h_OutLocArry, 2 * MaxPointNum * OutParaNumGS2D * sizeof(float));

	hipMalloc((void **)&d_LocArry_LastGroup, MaxPointNum * OutParaNumGS2D * sizeof(float));
	
	hipMalloc((void **)&d_LocArry_ConsecFit, 2 * MaxPointNum * OutParaNumGS2D * sizeof(float));


	// consecutive molecule find

	hipMalloc((void **)&d_ForwardLinkID, 2 * MaxPointNum * sizeof(int));
	hipMalloc((void **)&d_BackwardLinkID, 2 * MaxPointNum * sizeof(int));


	// on time calculate

	hipHostMalloc((void **)&h_OntimeDistrib, MaxOnTimeConsecutiveNum * sizeof(int));
	hipMalloc((void **)&d_OntimeDistrib, MaxOnTimeConsecutiveNum * sizeof(int));

	hipHostMalloc((void **)&h_ValidFluoNum, sizeof(int));
	hipMalloc((void **)&d_ValidFluoNum, sizeof(int));

	h_OntimeRatio = new float[MaxOnTimeConsecutiveNum];

}

void ConsecutiveFit_TypeDef::Deinit()
{
	hipHostFree(h_OutLocArry);

	hipFree(d_LocArry_LastGroup);

	hipFree(d_LocArry_ConsecFit);

	// consecutive molecule find
	hipFree(d_ForwardLinkID);
	hipFree(d_BackwardLinkID);


	// on time calculate

	hipHostFree(h_OntimeDistrib);
	hipFree(d_OntimeDistrib);

	hipHostFree(h_ValidFluoNum);
	hipFree(d_ValidFluoNum);

	delete[]h_OntimeRatio;
}



