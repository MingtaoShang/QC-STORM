/*
This program is free software: you can redistribute it and/or modify
it under the terms of the GNU LESSER GENERAL PUBLIC LICENSE as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU LESSER GENERAL PUBLIC LICENSE for more details.

You should have received a copy of the GNU LESSER GENERAL PUBLIC LICENSE
along with this program.  If not, see <https://www.gnu.org/licenses/>.
*/


#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>


#include "bfgsMLE_GS2D_2Emitter_core.h"

#include "bfgsMLE_GS2D_3Emitter_core.h"

#include "bfgsMLE_AS3D_2Emitter_core.h"


#include "bfgsMLE_Multi_Emitter_Fit.h"



void HDLoc_BFGS_MLELocalization_2D_2Emitter(float * d_LocArry, unsigned short * d_ImageROI, float *d_WLEPara, int MultiFitFluoNum_2E, FitPosInf_TypeDef* d_FitPosInf, CoreFittingPara *d_FitPara, int ROISize, hipStream_t cstream)
{
	hipError_t err;


	int BlockDim = ThreadsPerBlock;
	int BlockNum = ((MultiFitFluoNum_2E + ThreadsPerBlock - 1) / ThreadsPerBlock);
	
	
	// MLEROILocTop execute
	switch (ROISize)
	{	
	case 5:
		bfgsMLELoc_Gauss2D_2E<5, FitParaNum_2D_2E> << <BlockNum, BlockDim, 0, cstream >> > (d_LocArry, d_ImageROI, d_WLEPara, MultiFitFluoNum_2E, d_FitPosInf, d_FitPara);
		break;
	case 7:
		bfgsMLELoc_Gauss2D_2E<7, FitParaNum_2D_2E> << <BlockNum, BlockDim, 0, cstream >> > (d_LocArry, d_ImageROI, d_WLEPara, MultiFitFluoNum_2E, d_FitPosInf, d_FitPara);
		break;
	case 9:
		bfgsMLELoc_Gauss2D_2E<9, FitParaNum_2D_2E> << <BlockNum, BlockDim, 0, cstream >> > (d_LocArry, d_ImageROI, d_WLEPara, MultiFitFluoNum_2E, d_FitPosInf, d_FitPara);
		break;
	case 11:
		bfgsMLELoc_Gauss2D_2E<11, FitParaNum_2D_2E> << <BlockNum, BlockDim, 0, cstream >> > (d_LocArry, d_ImageROI, d_WLEPara, MultiFitFluoNum_2E, d_FitPosInf, d_FitPara);
		break;
	case 13:
		bfgsMLELoc_Gauss2D_2E<13, FitParaNum_2D_2E> << <BlockNum, BlockDim, 0, cstream >> > (d_LocArry, d_ImageROI, d_WLEPara, MultiFitFluoNum_2E, d_FitPosInf, d_FitPara);
		break;
	
		/*
	// shared memory is not enough
	case 15:

	break;
	case 17:

	break;
		*/
	default:

		break;
	}

	hipStreamQuery(cstream);

}



void HDLoc_BFGS_MLELocalization_2D_3Emitter(float * d_LocArry, unsigned short * d_ImageROI, float *d_WLEPara, int MultiFitFluoNum_3E, FitPosInf_TypeDef* d_FitPosInf, CoreFittingPara *d_FitPara, int ROISize, hipStream_t cstream)
{
	hipError_t err;


	int BlockDim = ThreadsPerBlock;
	int BlockNum = ((MultiFitFluoNum_3E + ThreadsPerBlock - 1) / ThreadsPerBlock);


	// MLEROILocTop execute
	switch (ROISize)
	{
	case 5:
		bfgsMLELoc_Gauss2D_3E<5, FitParaNum_2D_3E> << <BlockNum, BlockDim, 0, cstream >> > (d_LocArry, d_ImageROI, d_WLEPara, MultiFitFluoNum_3E, d_FitPosInf, d_FitPara);
		break;
	case 7:
		bfgsMLELoc_Gauss2D_3E<7, FitParaNum_2D_3E> << <BlockNum, BlockDim, 0, cstream >> > (d_LocArry, d_ImageROI, d_WLEPara, MultiFitFluoNum_3E, d_FitPosInf, d_FitPara);
		break;
	case 9:
		bfgsMLELoc_Gauss2D_3E<9, FitParaNum_2D_3E> << <BlockNum, BlockDim, 0, cstream >> > (d_LocArry, d_ImageROI, d_WLEPara, MultiFitFluoNum_3E, d_FitPosInf, d_FitPara);
		break;
	case 11:
		bfgsMLELoc_Gauss2D_3E<11, FitParaNum_2D_3E> << <BlockNum, BlockDim, 0, cstream >> > (d_LocArry, d_ImageROI, d_WLEPara, MultiFitFluoNum_3E, d_FitPosInf, d_FitPara);
		break;
	case 13:
		bfgsMLELoc_Gauss2D_3E<13, FitParaNum_2D_3E> << <BlockNum, BlockDim, 0, cstream >> > (d_LocArry, d_ImageROI, d_WLEPara, MultiFitFluoNum_3E, d_FitPosInf, d_FitPara);
		break;

		/*
		// shared memory is not enough
		case 15:

		break;
		case 17:

		break;
		*/
	default:

		break;
	}

	hipStreamQuery(cstream);

}



void HDLoc_BFGS_MLELocalization_AS3D_2Emitter(float * d_LocArry, unsigned short * d_ImageROI, float *d_WLEPara, int MultiFitFluoNum_2E, FitPosInf_TypeDef* d_FitPosInf, CoreFittingPara *d_FitPara, int ROISize, hipStream_t cstream)
{
	hipError_t err;


	int BlockDim = ThreadsPerBlock;
	int BlockNum = ((MultiFitFluoNum_2E + ThreadsPerBlock - 1) / ThreadsPerBlock);


	// MLEROILocTop_AS3D execute
	switch (ROISize)
	{

	case 5:
		bfgsMLELoc_AS3D_2E<5, FitParaNum_AS3D_2E> << <BlockNum, BlockDim, 0, cstream >> >(d_LocArry, d_ImageROI, d_WLEPara, MultiFitFluoNum_2E, d_FitPosInf, d_FitPara);
		break;
	case 7:
		bfgsMLELoc_AS3D_2E<7, FitParaNum_AS3D_2E> << <BlockNum, BlockDim, 0, cstream >> >(d_LocArry, d_ImageROI, d_WLEPara, MultiFitFluoNum_2E, d_FitPosInf, d_FitPara);
		break;
	case 9:
		bfgsMLELoc_AS3D_2E<8, FitParaNum_AS3D_2E> << <BlockNum, BlockDim, 0, cstream >> >(d_LocArry, d_ImageROI, d_WLEPara, MultiFitFluoNum_2E, d_FitPosInf, d_FitPara);
		break;
	case 11:
		bfgsMLELoc_AS3D_2E<11, FitParaNum_AS3D_2E> << <BlockNum, BlockDim, 0, cstream >> >(d_LocArry, d_ImageROI, d_WLEPara, MultiFitFluoNum_2E, d_FitPosInf, d_FitPara);
		break;
	case 13:
		bfgsMLELoc_AS3D_2E<13, FitParaNum_AS3D_2E> << <BlockNum, BlockDim, 0, cstream >> >(d_LocArry, d_ImageROI, d_WLEPara, MultiFitFluoNum_2E, d_FitPosInf, d_FitPara);
		break;

		/*
		// shared memory is not enough
		case 15:

		break;
		case 17:

		break;

		*/

	default:

		break;
	}

	hipStreamQuery(cstream);

}
