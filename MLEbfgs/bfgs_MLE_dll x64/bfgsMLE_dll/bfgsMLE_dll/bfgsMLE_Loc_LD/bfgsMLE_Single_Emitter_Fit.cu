/*
This program is free software: you can redistribute it and/or modify
it under the terms of the GNU LESSER GENERAL PUBLIC LICENSE as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU LESSER GENERAL PUBLIC LICENSE for more details.

You should have received a copy of the GNU LESSER GENERAL PUBLIC LICENSE
along with this program.  If not, see <https://www.gnu.org/licenses/>.
*/


#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>


#include "bfgsMLE_GS2D_core.h"

#include "bfgsMLE_AS3D_core.h"





void LDLoc_BFGS_MLELocalizationGS2D(float * d_LocArry, unsigned short * d_ImageROI, float *d_WLEPara, int SingleFitNum, FitPosInf_TypeDef* d_FitPosInf, CoreFittingPara *d_FitPara, int ROISize, hipStream_t cstream)
{
	hipError_t err;

	int BlockDim = ThreadsPerBlock;
	int BlockNum = ((SingleFitNum + ThreadsPerBlock - 1) / ThreadsPerBlock);
	

	// MLEROILocTop execute
	switch (ROISize)
	{	
	case 5:
		bfgsMLELoc_Gauss2D<5, FitParaNum_2D> << <BlockNum, BlockDim, 0, cstream >> > (d_LocArry, d_ImageROI, d_WLEPara, SingleFitNum, d_FitPosInf, d_FitPara);
		break;
	case 7:
		bfgsMLELoc_Gauss2D<7, FitParaNum_2D> << <BlockNum, BlockDim, 0, cstream >> > (d_LocArry, d_ImageROI, d_WLEPara, SingleFitNum, d_FitPosInf, d_FitPara);
		break;
	case 9:
		bfgsMLELoc_Gauss2D<9, FitParaNum_2D> << <BlockNum, BlockDim, 0, cstream >> > (d_LocArry, d_ImageROI, d_WLEPara, SingleFitNum, d_FitPosInf, d_FitPara);
		break;
	case 11:
		bfgsMLELoc_Gauss2D<11, FitParaNum_2D> << <BlockNum, BlockDim, 0, cstream >> > (d_LocArry, d_ImageROI, d_WLEPara, SingleFitNum, d_FitPosInf, d_FitPara);
		break;
	case 13:
		bfgsMLELoc_Gauss2D<13, FitParaNum_2D> << <BlockNum, BlockDim, 0, cstream >> > (d_LocArry, d_ImageROI, d_WLEPara, SingleFitNum, d_FitPosInf, d_FitPara);
		break;
	case 15:
		bfgsMLELoc_Gauss2D<15, FitParaNum_2D> << <BlockNum, BlockDim, 0, cstream >> > (d_LocArry, d_ImageROI, d_WLEPara, SingleFitNum, d_FitPosInf, d_FitPara);
		break;
	case 17:
		bfgsMLELoc_Gauss2D<17, FitParaNum_2D> << <BlockNum, BlockDim, 0, cstream >> > (d_LocArry, d_ImageROI, d_WLEPara, SingleFitNum, d_FitPosInf, d_FitPara);
		break;

	default:

		break;
	}

	hipStreamQuery(cstream);

}




void LDLoc_BFGS_MLELocalizationAS3D(float * d_LocArry, unsigned short * d_ImageROI, float *d_WLEPara, int SingleFitNum, FitPosInf_TypeDef* d_FitPosInf, CoreFittingPara *d_FitPara, int ROISize, hipStream_t cstream)
{
	hipError_t err;


	int BlockDim = ThreadsPerBlock;
	int BlockNum = ((SingleFitNum + ThreadsPerBlock - 1) / ThreadsPerBlock);


	// MLEROILocTop_AS3D execute
	switch (ROISize)
	{

	case 5:
		bfgsMLELoc_AS3D<5, FitParaNum_AS3D> << <BlockNum, BlockDim, 0, cstream >> >(d_LocArry, d_ImageROI, d_WLEPara, SingleFitNum, d_FitPosInf, d_FitPara);
		break;
	case 7:
		bfgsMLELoc_AS3D<7, FitParaNum_AS3D> << <BlockNum, BlockDim, 0, cstream >> >(d_LocArry, d_ImageROI, d_WLEPara, SingleFitNum, d_FitPosInf, d_FitPara);
		break;
	case 9:
		bfgsMLELoc_AS3D<8, FitParaNum_AS3D> << <BlockNum, BlockDim, 0, cstream >> >(d_LocArry, d_ImageROI, d_WLEPara, SingleFitNum, d_FitPosInf, d_FitPara);
		break;
	case 11:
		bfgsMLELoc_AS3D<11, FitParaNum_AS3D> << <BlockNum, BlockDim, 0, cstream >> >(d_LocArry, d_ImageROI, d_WLEPara, SingleFitNum, d_FitPosInf, d_FitPara);
		break;
	case 13:
		bfgsMLELoc_AS3D<13, FitParaNum_AS3D> << <BlockNum, BlockDim, 0, cstream >> >(d_LocArry, d_ImageROI, d_WLEPara, SingleFitNum, d_FitPosInf, d_FitPara);
		break;
	case 15:
		bfgsMLELoc_AS3D<15, FitParaNum_AS3D> << <BlockNum, BlockDim, 0, cstream >> >(d_LocArry, d_ImageROI, d_WLEPara, SingleFitNum, d_FitPosInf, d_FitPara);
		break;
	case 17:
		bfgsMLELoc_AS3D<17, FitParaNum_AS3D> << <BlockNum, BlockDim, 0, cstream >> >(d_LocArry, d_ImageROI, d_WLEPara, SingleFitNum, d_FitPosInf, d_FitPara);
		break;


	default:

		break;
	}

	hipStreamQuery(cstream);

}
