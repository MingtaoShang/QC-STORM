#include "hip/hip_runtime.h"
/*
This program is free software: you can redistribute it and/or modify
it under the terms of the GNU LESSER GENERAL PUBLIC LICENSE as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU LESSER GENERAL PUBLIC LICENSE for more details.

You should have received a copy of the GNU LESSER GENERAL PUBLIC LICENSE
along with this program.  If not, see <https://www.gnu.org/licenses/>.
*/

#include "bfgs_top.h"

#include "bfgsMLE_core.h"


#include "bfgs_LocalizationFilter.h"
#include "bfgs_OntimeCalc.h"

#include "LDROIExtraction.h"


void LDLocData_TypeDef::BFGS_MLELocalization(unsigned short * h_ImageROI, float *h_WLEPara, LocalizationPara & LocPara, int FluoNum, hipStream_t cstream)
{
	// note there are zeros points in the localization results, there are filtered failed results,
	// they should not be classified into false positive or negative in algorithm evaluation

	hipError_t err;

	int LocType = LocPara.LocType;
	int ROISize = LocPara.ROISize;

	int ROIWholeSize = (ROISize*(ROISize + 1));

	oValidFluoNum = FluoNum;

	err = hipMemcpyAsync(d_ImageROI, h_ImageROI, FluoNum * ROIWholeSize *sizeof(short), hipMemcpyHostToDevice, cstream);

	if (WLE_ENABLE)
	{
		hipMemcpyAsync(d_WLEPara, h_WLEPara, FluoNum * WLE_ParaNumber * sizeof(float), hipMemcpyHostToDevice, cstream);
	}

	// for multi emitter fitting
	hipMemsetAsync(d_MultiFitFluoNum, 0, sizeof(int), cstream);


	// Low density fitting
	if (LocPara.LocType == LocType_GS2D)
	{
		// for 2d round Gaussian localization
		LDLoc_BFGS_MLELocalizationGS2D(d_LocArry, d_ImageROI, d_WLEPara, d_MultiFitFluoNum, d_MultiFitFluoPos, LocPara, FluoNum, cstream);
	
	}
	else if (LocPara.LocType == LocType_AS3D)
	{
		// for 3d astigmatism elliptical Gaussian localization
		LDLoc_BFGS_MLELocalizationAS3D(d_LocArry, d_ImageROI, d_WLEPara, d_MultiFitFluoNum, d_MultiFitFluoPos, LocPara, FluoNum, cstream);
	
	}

	// high density fitting
	if (LocPara.MultiEmitterFitEn)
	{
		hipMemcpyAsync(h_MultiFitFluoNum, d_MultiFitFluoNum, sizeof(int), hipMemcpyDeviceToHost, cstream);
		hipStreamSynchronize(cstream);

		int MultiFitFluoNum = *h_MultiFitFluoNum;

		MultiFitRatio = (float)MultiFitFluoNum / FluoNum;

		if (LocPara.LocType == LocType_GS2D)
		{
			HDLoc_BFGS_MLELocalization_2D_2Emitter(d_LocArry, d_ImageROI, d_WLEPara, MultiFitFluoNum, d_MultiFitFluoPos, LocPara, cstream);
		
		}
		else if (LocPara.LocType == LocType_AS3D)
		{
			HDLoc_BFGS_MLELocalization_AS3D_2Emitter(d_LocArry, d_ImageROI, d_WLEPara, MultiFitFluoNum, d_MultiFitFluoPos, LocPara, cstream);
		
		}

		hipStreamSynchronize(cstream);
	}




	// localization precision calculated by CRLB
	LocPrecCalc_GaussianCRLB(d_LocArry, LocPara, FluoNum, cstream);


	// filter bad molecules
	FilterBadFit(LocPara, FluoNum, cstream);


	// 2d and AS3d
	hipMemcpyAsync(h_LocArry, d_LocArry, oValidFluoNum*OutParaNumGS2D * sizeof(float), hipMemcpyDeviceToHost, cstream);
	hipStreamSynchronize(cstream);


}


void LDLocData_TypeDef::Init(LocalizationPara & LocPara)
{

	hipError_t err;

	int ROIWholeSize = LocPara.ROISize*(LocPara.ROISize + 1);


	// host and gpu
	err = hipHostMalloc((void **)&h_ImageROI, MaxPointNum*ROIWholeSize*sizeof(short));
	HandleErr(err, "hipHostMalloc LDLoc h_ImageROI");

	err = hipMalloc((void **)&d_ImageROI, MaxPointNum*ROIWholeSize*sizeof(short));
	HandleErr(err, "hipMalloc LDLoc d_ImageROI");

	hipHostMalloc((void **)&h_LocArry, MaxPointNum*OutParaNumAS3D*sizeof(float));
	hipMalloc((void **)&d_LocArry, MaxPointNum*OutParaNumAS3D*sizeof(float));


	err = hipMalloc((void **)&d_WLEPara, MaxPointNum * WLE_ParaNumber * sizeof(float));


	// Consecutive fitting from adjecent frames

	hipMalloc((void **)&d_ForwardLinkID, MaxPointNum*sizeof(int));
	hipMalloc((void **)&d_BackwardLinkID, MaxPointNum*sizeof(int));
	hipMalloc((void **)&d_ConsecutiveNum, MaxPointNum*sizeof(int));


	hipHostMalloc((void **)&h_OntimeDistrib, MaxOnTimeConsecutiveNum*sizeof(int));
	hipMalloc((void **)&d_OntimeDistrib, MaxOnTimeConsecutiveNum*sizeof(int));

	hipHostMalloc((void **)&h_ValidFluoNum, sizeof(int));
	hipMalloc((void **)&d_ValidFluoNum, sizeof(int));

	h_OntimeRatio = new float[MaxOnTimeConsecutiveNum];


	// for loc filter
	err = hipHostMalloc((void **)&h_SNRSumUp, sizeof(float));
	err = hipHostMalloc((void **)&h_ValidNum, sizeof(int));

	hipMalloc((void **)&d_SNRSumUp, sizeof(float));
	hipMalloc((void **)&d_ValidNum, sizeof(int));


	// multi emitter fitting
	hipHostMalloc((void **)&h_MultiFitFluoNum, sizeof(int));
	hipMalloc((void **)&d_MultiFitFluoNum, sizeof(int));

	hipMalloc((void **)&d_MultiFitFluoPos, MaxPointNum * sizeof(int));

	MultiFitRatio = 0;

}

void LDLocData_TypeDef::Deinit( LocalizationPara & LocPara)
{
	hipError_t err;

	err = hipHostFree(h_ImageROI);
	HandleErr(err, "hipHostFree LDLoc h_ImageROI");
	err = hipFree(d_ImageROI);
	HandleErr(err, "hipFree LDLoc d_ImageROI");

	err = hipHostFree(h_LocArry);
	err = hipFree(d_LocArry);

	err = hipFree(d_WLEPara);
	// Consecutive fitting from adjecent frames

	hipFree(d_ForwardLinkID);
	hipFree(d_BackwardLinkID);
	hipFree(d_ConsecutiveNum);

	hipHostFree(h_OntimeDistrib);
	hipFree(d_OntimeDistrib);

	hipHostFree(h_ValidFluoNum);
	hipFree(d_ValidFluoNum);

	delete[](h_OntimeRatio);


	// for loc filter
	hipHostFree(h_SNRSumUp);
	hipHostFree(h_ValidNum);

	hipFree(d_SNRSumUp);
	hipFree(d_ValidNum);



	// multi emitter fitting
	hipHostFree(h_MultiFitFluoNum);

	hipFree(d_MultiFitFluoNum);
	hipFree(d_MultiFitFluoPos);


}


int LDLocData_TypeDef::GetFirstFrame(float * h_LocArry, int FluoNum)
{
	float(*pLocArry)[OutParaNumGS2D]; // for parameter array

	pLocArry = (float(*)[OutParaNumGS2D])h_LocArry;

	int cnt = 0;
	int curFrame = 0;
	for (cnt = 0; cnt < FluoNum; cnt++)
	{
		curFrame = pLocArry[cnt][Pos_Frme];

		if (curFrame != 0)break;
	}


	return curFrame;
}

int LDLocData_TypeDef::GetLastFrame(float * h_LocArry, int FluoNum)
{
	float(*pLocArry)[OutParaNumGS2D]; // for parameter array

	pLocArry = (float(*)[OutParaNumGS2D])h_LocArry;

	int cnt = 0;
	int curFrame = 0;
	for (cnt = FluoNum - 1; cnt > 0; cnt--)
	{
		curFrame = pLocArry[cnt][Pos_Frme];

		if (curFrame != 0)break;
	}


	return curFrame;
}
int LDLocData_TypeDef::GetFirstFrameFromROI(unsigned short * h_ImageROI, int ROISize, int FluoNum)
{

	int WholeROISize = ROISize*(ROISize + 1);
	int ROIPixelSize = ROISize*ROISize;

	int FrameAddrBias = ROIPixelSize + 2;
	int curFrame = h_ImageROI[FrameAddrBias];

	return curFrame;
}

int LDLocData_TypeDef::GetLastFrameFromROI(unsigned short * h_ImageROI, int ROISize, int FluoNum)
{
	int WholeROISize = ROISize*(ROISize + 1);
	int ROIPixelSize = ROISize*ROISize;

	int FrameAddrBias = WholeROISize*(FluoNum - 2) + ROIPixelSize + 2;
	int curFrame = h_ImageROI[FrameAddrBias];

	return curFrame;
}

