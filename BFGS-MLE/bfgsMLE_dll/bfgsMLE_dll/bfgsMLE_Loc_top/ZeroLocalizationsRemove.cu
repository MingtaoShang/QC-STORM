#include "hip/hip_runtime.h"
#include "ZeroLocalizationsRemove.h"




void ZeroLocalizationsRemovel_TypeDef::RemoveZeroLocalizations(float *ih_LocArry, int iFluoNum, hipStream_t cstream)
{
	FindCopyID(ih_LocArry, iFluoNum);


	hipMemcpyAsync(d_LocArry_Raw, ih_LocArry, iFluoNum * OutParaNumGS2D * sizeof(float), hipMemcpyHostToDevice, cstream);
	hipMemcpyAsync(d_FluoID_Valid, h_FluoID_Valid, ValidFluoNum * sizeof(int), hipMemcpyHostToDevice, cstream);

	int BlockDim = ThreadsPerBlock;
	int BlockNum = ((ValidFluoNum + ThreadsPerBlock - 1) / ThreadsPerBlock);

	gpuRemoveZeorLocalizations << <BlockNum, BlockDim, 0, cstream >> > (d_LocArry_Valid, d_LocArry_Raw, d_FluoID_Valid, ValidFluoNum);

	hipMemcpyAsync(h_LocArry, d_LocArry_Valid, ValidFluoNum * OutParaNumGS2D * sizeof(float), hipMemcpyDeviceToHost, cstream);

	hipStreamSynchronize(cstream);

}

void ZeroLocalizationsRemovel_TypeDef::FindCopyID(float *ih_LocArry, int iFluoNum)
{
	float(*pLocArry)[OutParaNumGS2D] = (float(*)[OutParaNumGS2D])ih_LocArry;

	int ValidId = 0;
	float PeakPhoton = 0;

	for (int CurID = 0; CurID < iFluoNum; CurID++)
	{
		PeakPhoton = pLocArry[CurID][Pos_PPho];

		if (PeakPhoton > 1.0f)
		{
			h_FluoID_Valid[ValidId] = CurID;
			ValidId++;
		}
	}

	ValidFluoNum = ValidId;

//	printf("ValidFluoNum:%d \n", ValidFluoNum);
}

void ZeroLocalizationsRemovel_TypeDef::Init()
{
	ValidFluoNum = 0;


	// host and gpu
	hipHostMalloc((void **)&h_LocArry, MaxPointNum*OutParaNumGS2D * sizeof(float));

	hipMalloc((void **)&d_LocArry_Raw, MaxPointNum*OutParaNumGS2D * sizeof(float));
	hipMalloc((void **)&d_LocArry_Valid, MaxPointNum*OutParaNumGS2D * sizeof(float));

	hipHostMalloc((void **)&h_FluoID_Valid, MaxPointNum * sizeof(int));

	hipMalloc((void **)&d_FluoID_Valid, MaxPointNum * sizeof(int));

}

void ZeroLocalizationsRemovel_TypeDef::Deinit()
{
	hipHostFree(h_LocArry);

	hipFree(d_LocArry_Raw);
	hipFree(d_LocArry_Valid);

	hipHostFree(h_FluoID_Valid);

	hipFree(d_FluoID_Valid);

}



__global__ void gpuRemoveZeorLocalizations(float * d_LocArry_Valid, float * d_LocArry_Raw, int *d_FluoID_Valid, int ValidFluoNum)
{
	int gid = blockDim.x*blockIdx.x + threadIdx.x;

	float(*pLocArry_Valid)[OutParaNumGS2D] = (float(*)[OutParaNumGS2D])d_LocArry_Valid;
	float(*pLocArry_Raw)[OutParaNumGS2D] = (float(*)[OutParaNumGS2D])d_LocArry_Raw;

	if (gid < ValidFluoNum)
	{
		int RawDataID = d_FluoID_Valid[gid];

#pragma unroll
		for (int cnt = 0; cnt < OutParaNumGS2D; cnt++)
		{
			pLocArry_Valid[gid][cnt] = pLocArry_Raw[RawDataID][cnt];
		}
	}
}

