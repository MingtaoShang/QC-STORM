#include "hip/hip_runtime.h"
/*
This program is free software: you can redistribute it and/or modify
it under the terms of the GNU LESSER GENERAL PUBLIC LICENSE as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU LESSER GENERAL PUBLIC LICENSE for more details.

You should have received a copy of the GNU LESSER GENERAL PUBLIC LICENSE
along with this program.  If not, see <https://www.gnu.org/licenses/>.
*/

#include "BackgroundNoiseFilter.h"


void BackgroundNoiseFilter_TypeDef::FilterBackgroundNoise(float * h_iLocArry, int FluoNum, int DataSource, hipStream_t cstream)
{

	int BlockDim = ThreadsPerBlock;
	int BlockNum = (FluoNum + ThreadsPerBlock - 1) / ThreadsPerBlock;

	if (DataSource == ImageSource_CPU_Pinned)
	{
		hipMemcpyAsync(d_LocArry, h_iLocArry, FluoNum*OutParaNumGS2D * sizeof(float), hipMemcpyHostToDevice, cstream);
	}
	else if(DataSource == ImageSource_CPU_Normal)
	{
		hipMemcpy(d_LocArry, h_iLocArry, FluoNum*OutParaNumGS2D * sizeof(float), hipMemcpyHostToDevice);
	}
	else if (DataSource == ImageSource_GPU)
	{
		hipMemcpyAsync(d_LocArry, h_iLocArry, FluoNum*OutParaNumGS2D * sizeof(float), hipMemcpyDeviceToDevice, cstream);
	}
	else
	{
		return;
	}


	BackgroundNoiseRemove(FluoNum, cstream);

	if (DataSource == ImageSource_CPU_Pinned)
	{
		hipMemcpyAsync(h_LocArry, d_LocArry, FluoNum*OutParaNumGS2D * sizeof(float), hipMemcpyDeviceToHost, cstream);
	}
	else if (DataSource == ImageSource_CPU_Normal)
	{
		hipMemcpy(h_LocArry, d_LocArry, FluoNum*OutParaNumGS2D * sizeof(float), hipMemcpyDeviceToHost);
	}


	hipStreamSynchronize(cstream);

}

void BackgroundNoiseFilter_TypeDef::BackgroundNoiseRemove(int FluoNum, hipStream_t cstream)
{
	int BlockDim = ThreadsPerBlock;
	int BlockNum = (FluoNum + ThreadsPerBlock - 1) / ThreadsPerBlock;

	NeighborNumCalc(FluoNum, cstream);

	gpuNoiseIdenfity_NoiseFilter << <BlockNum, BlockDim, 0, cstream >> > (d_IsNoise, d_NeighborNum_Th1, d_NeighborNum_Th2, d_NeighborNum_Th3, FluoNum);
	gpuRemoveNoiseFluo_NoiseFilter << <BlockNum, BlockDim, 0, cstream >> > (d_LocArry, d_IsNoise, FluoNum);

	hipStreamSynchronize(cstream);

}

void BackgroundNoiseFilter_TypeDef::NeighborNumCalc(int FluoNum, hipStream_t cstream)
{
	int BlockDim = ThreadsPerBlock;
	int BlockNum = (FluoNum + ThreadsPerBlock - 1) / ThreadsPerBlock;

	float MeanDistance = GetMeanMinNearestNeighborDistance(FluoNum, cstream);

	float Distance_Th1 = MeanDistance * 4.0f;
	float Distance_Th2 = Distance_Th1 * 2.5;
	float Distance_Th3 = Distance_Th1 * 5;

	hipMemsetAsync(d_NeighborNum_Th1, 0, FluoNum * sizeof(int), cstream);
	hipMemsetAsync(d_NeighborNum_Th2, 0, FluoNum * sizeof(int), cstream);
	hipMemsetAsync(d_NeighborNum_Th3, 0, FluoNum * sizeof(int), cstream);

	gpuNeighborNumberCalc_NoiseFilter << <BlockNum, BlockDim, 0, cstream >> > (d_LocArry, d_NeighborNum_Th1, d_NeighborNum_Th2, d_NeighborNum_Th3, Distance_Th1, Distance_Th2, Distance_Th3, FluoNum);

	hipStreamSynchronize(cstream);

}


float BackgroundNoiseFilter_TypeDef::GetMeanMinNearestNeighborDistance(int FluoNum, hipStream_t cstream)
{
	// get min nearest neighboring distance
	int SelFluoNum = min(25000, FluoNum); // 

	GetMinNearestNeighborDistance(FluoNum, SelFluoNum, cstream);

	int BlockDim = ThreadsPerBlock;
	int BlockNum = (SelFluoNum + ThreadsPerBlock - 1) / ThreadsPerBlock;

	// get mean min nearest neighboring distance

	// calculate mean distance again without unnormal molecules filtered
	hipMemsetAsync(d_ValidNum, 0, sizeof(float), cstream);
	hipMemsetAsync(d_TotalValue, 0, sizeof(int), cstream);

	float Distance_Th_Pixel = 400;
	gpuMeanDistanceCalc_NoiseFilter << <BlockNum, BlockDim, 0, cstream >> > (d_MinDistance, d_ValidNum, d_TotalValue, Distance_Th_Pixel, SelFluoNum);

	//
	hipMemcpyAsync(h_ValidNum, d_ValidNum,  sizeof(float), hipMemcpyDeviceToHost, cstream);
	hipMemcpyAsync(h_TotalValue, d_TotalValue, sizeof(float), hipMemcpyDeviceToHost, cstream);
	hipStreamSynchronize(cstream);

	if (*h_ValidNum < 1)*h_ValidNum = 1;
	float MeanDistance = *h_TotalValue / *h_ValidNum;

//	printf("mean distance:%f %f %f\n", MeanDistance, *h_TotalValue, *h_ValidNum);

	// calculate mean distance again with unnormal molecules filtered
	hipMemsetAsync(d_ValidNum, 0, sizeof(float), cstream);
	hipMemsetAsync(d_TotalValue, 0, sizeof(int), cstream);

	Distance_Th_Pixel = MeanDistance * 4;
	gpuMeanDistanceCalc_NoiseFilter << <BlockNum, BlockDim, 0, cstream >> > (d_MinDistance, d_ValidNum, d_TotalValue, Distance_Th_Pixel, SelFluoNum);

	//
	hipMemcpyAsync(h_ValidNum, d_ValidNum, sizeof(float), hipMemcpyDeviceToHost, cstream);
	hipMemcpyAsync(h_TotalValue, d_TotalValue, sizeof(float), hipMemcpyDeviceToHost, cstream);
	hipStreamSynchronize(cstream);

	if (*h_ValidNum < 1)*h_ValidNum = 1;
	MeanDistance = *h_TotalValue / *h_ValidNum;

//	printf("mean distance:%f %f %f\n", MeanDistance, *h_TotalValue, *h_ValidNum);

	return MeanDistance;

}


void BackgroundNoiseFilter_TypeDef::GetMinNearestNeighborDistance(int FluoNum, int SelFluoNum, hipStream_t cstream)
{
	SelFluoNum = min(22000, FluoNum);

	int BlockDim = ThreadsPerBlock;
	int BlockNum = (SelFluoNum + ThreadsPerBlock - 1) / ThreadsPerBlock;

	hipMemsetAsync(d_MinDistance, 0, SelFluoNum * sizeof(float), cstream);

	gpuMinDistanceCalc_NoiseFilter << <BlockNum, BlockDim, 0, cstream >> > (d_LocArry, d_MinDistance, SelFluoNum, FluoNum);
	hipStreamSynchronize(cstream);

}



void BackgroundNoiseFilter_TypeDef::Init(int TotalFluoNum)
{
	hipHostMalloc((void **)&h_LocArry, TotalFluoNum * OutParaNumGS2D*sizeof(float));
	hipMalloc((void **)&d_LocArry, TotalFluoNum * OutParaNumGS2D*sizeof(float));

	//
	hipMalloc((void **)&d_MinDistance, TotalFluoNum * sizeof(float));

	hipHostMalloc((void **)&h_ValidNum, sizeof(float));
	hipHostMalloc((void **)&h_TotalValue, sizeof(float));

	hipMalloc((void **)&d_ValidNum,   sizeof(float));
	hipMalloc((void **)&d_TotalValue,   sizeof(float));

	//
	hipMalloc((void **)&d_NeighborNum_Th1, TotalFluoNum * sizeof(int));
	hipMalloc((void **)&d_NeighborNum_Th2, TotalFluoNum * sizeof(int));
	hipMalloc((void **)&d_NeighborNum_Th3, TotalFluoNum * sizeof(int));

	hipMalloc((void **)&d_IsNoise, TotalFluoNum * sizeof(int));

}


void BackgroundNoiseFilter_TypeDef::DeInit()
{
	hipHostFree(h_LocArry);
	hipFree(d_LocArry);
	//
	hipFree(d_MinDistance);

	hipHostFree(h_ValidNum);
	hipHostFree(h_TotalValue);

	hipFree(d_ValidNum);
	hipFree(d_TotalValue);

	//
	hipFree(d_NeighborNum_Th1);
	hipFree(d_NeighborNum_Th2);
	hipFree(d_NeighborNum_Th3);

	hipFree(d_IsNoise);

}


