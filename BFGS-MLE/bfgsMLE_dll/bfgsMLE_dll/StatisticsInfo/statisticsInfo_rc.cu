#include "hip/hip_runtime.h"
/*
This program is free software: you can redistribute it and/or modify
it under the terms of the GNU LESSER GENERAL PUBLIC LICENSE as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU LESSER GENERAL PUBLIC LICENSE for more details.

You should have received a copy of the GNU LESSER GENERAL PUBLIC LICENSE
along with this program.  If not, see <https://www.gnu.org/licenses/>.
*/

#include "statisticsInfo.h"
#include "cudaWrapper.h"


float FluoStatisticData_TypeDef::GetActivationDensity(float Ov1MoleculesRatio, float RadiusTh_um)
{
	float density = 0;
	// some times when image is very small and molecular is very little, the ratio can be 1, lead the density to be 0
	if (Ov1MoleculesRatio <= 0.001f)Ov1MoleculesRatio = 0.001f;
	if (Ov1MoleculesRatio > 1)Ov1MoleculesRatio = 1;

	float ratio1 = logf(Ov1MoleculesRatio);
	float pi = 3.14159f;

	density = -ratio1 / pi / RadiusTh_um / RadiusTh_um;

	// a compensation from practical to theoretical
	density = density*density* 0.3156f + density* 1.186f; // second order polynomial compensation

	return density;
}


float FluoStatisticData_TypeDef::GetHistogramMeanData(int *HistData, int DatLen, float PercentTh)
{
	//PercentTh=0.05; or 0.1
	int MaxDistribVal = 0;

	float CurValue = 0;

	float WeightedSum = 0;
	float OrdinarySum = 0;
	float MeanDat = 0;
	int cnt;

	MaxDistribVal = GetHistogramMaxData(HistData, DatLen);
	PercentTh = PercentTh*MaxDistribVal;

	// get mean by center of mass

	for (cnt = 0; cnt < DatLen; cnt++)
	{
		CurValue = HistData[cnt];

		if (CurValue >= PercentTh)
		{
			WeightedSum += CurValue * (cnt + 0.5f);
			OrdinarySum += CurValue;
		}
	}

	if (OrdinarySum == 0)OrdinarySum = 1;

	MeanDat = WeightedSum / OrdinarySum;

	return MeanDat;
}

int FluoStatisticData_TypeDef::GetHistogramMaxData(int *HistData, int DatLen)
{
	int cnt;
	int MaxDat = HistData[0];

	for (cnt = 0; cnt < DatLen; cnt++)
	{
		MaxDat = max(MaxDat, HistData[cnt]);
	}

	return MaxDat;
}

int FluoStatisticData_TypeDef::GetHistogramMaxDataPos(int *HistData, int DatLen)
{
	int cnt;
	int MaxPos = 0;
	float MaxDat = 0;

	for (cnt = 0; cnt < DatLen; cnt++)
	{
		if (MaxDat < HistData[cnt])
		{
			MaxDat = HistData[cnt];
			MaxPos = cnt;
		}
	}

	return MaxPos;
}

float FluoStatisticData_TypeDef::GetHistogramWidth(int *HistData, int MaxPos, int DatLen)
{
	// get gaussian distribution width (full width at half maximum)(2.2.3548f*sigma) by center of mass

	float WeightedSum = 0;
	float OrdinarySum = 0;

	float SigmaL;
	float SigmaR;
	int cnt = 0;

	if (MaxPos < 1)MaxPos = 1;
	if (MaxPos > DatLen - 1)MaxPos = DatLen - 1;



	WeightedSum = 0;
	OrdinarySum = 0;
	for (cnt = 1; cnt < MaxPos; cnt++)
	{
		WeightedSum += HistData[cnt] * (MaxPos - cnt);
		OrdinarySum += HistData[cnt];
	}
	SigmaL = WeightedSum / OrdinarySum;

	WeightedSum = 0;
	OrdinarySum = 0;
	for (cnt = MaxPos; cnt < DatLen - 1; cnt++)
	{
		WeightedSum += HistData[cnt] * (cnt - MaxPos);
		OrdinarySum += HistData[cnt];
	}
	SigmaR = WeightedSum / OrdinarySum;

	SigmaL = (SigmaL + SigmaR) / 2;

	return SigmaL;

}


