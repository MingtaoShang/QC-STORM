/*
This program is free software: you can redistribute it and/or modify
it under the terms of the GNU LESSER GENERAL PUBLIC LICENSE as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU LESSER GENERAL PUBLIC LICENSE for more details.

You should have received a copy of the GNU LESSER GENERAL PUBLIC LICENSE
along with this program.  If not, see <https://www.gnu.org/licenses/>.
*/


#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>



#include "bfgsMLE_AS3D_core.h"


#include "bfgs_top.h"



void LDLoc_BFGS_MLELocalizationAS3D( float * d_LocArry, unsigned short * d_SubRegion, float *d_WLEPara, LocalizationPara& LocPara, int FluoNum, hipStream_t cstream)
{
	hipError_t err;
	// MaxFluoNum must be the integer multiples of 32
	int ROISize = LocPara.ROISize;


	int BlockDim = ThreadsPerBlock;
	int BlockNum = ((FluoNum + ThreadsPerBlock - 1) / ThreadsPerBlock);


	// MLEROILocTop_AS3D execute
	switch (ROISize)
	{

	case 5:
		bfgsMLELoc_AS3D<5, FitParaNum_AS3D> << <BlockNum, BlockDim, 0, cstream >> >(d_LocArry, d_SubRegion, d_WLEPara, LocPara.Offset, LocPara.KAdc, LocPara.QE, LocPara.ZDepthCorrFactor, LocPara.p4, LocPara.p3, LocPara.p2, LocPara.p1, LocPara.p0, FluoNum);
		break;
	case 7:
		bfgsMLELoc_AS3D<7, FitParaNum_AS3D> << <BlockNum, BlockDim, 0, cstream >> >(d_LocArry, d_SubRegion, d_WLEPara, LocPara.Offset, LocPara.KAdc, LocPara.QE, LocPara.ZDepthCorrFactor, LocPara.p4, LocPara.p3, LocPara.p2, LocPara.p1, LocPara.p0, FluoNum);
		break;
	case 9:
		bfgsMLELoc_AS3D<8, FitParaNum_AS3D> << <BlockNum, BlockDim, 0, cstream >> >(d_LocArry, d_SubRegion, d_WLEPara, LocPara.Offset, LocPara.KAdc, LocPara.QE, LocPara.ZDepthCorrFactor, LocPara.p4, LocPara.p3, LocPara.p2, LocPara.p1, LocPara.p0, FluoNum);
		break;
	case 11:
		bfgsMLELoc_AS3D<11, FitParaNum_AS3D> << <BlockNum, BlockDim, 0, cstream >> >(d_LocArry, d_SubRegion, d_WLEPara, LocPara.Offset, LocPara.KAdc, LocPara.QE, LocPara.ZDepthCorrFactor, LocPara.p4, LocPara.p3, LocPara.p2, LocPara.p1, LocPara.p0, FluoNum);
		break;
	case 13:
		bfgsMLELoc_AS3D<13, FitParaNum_AS3D> << <BlockNum, BlockDim, 0, cstream >> >(d_LocArry, d_SubRegion, d_WLEPara, LocPara.Offset, LocPara.KAdc, LocPara.QE, LocPara.ZDepthCorrFactor, LocPara.p4, LocPara.p3, LocPara.p2, LocPara.p1, LocPara.p0, FluoNum);
		break;
	case 15:
		bfgsMLELoc_AS3D<15, FitParaNum_AS3D> << <BlockNum, BlockDim, 0, cstream >> >(d_LocArry, d_SubRegion, d_WLEPara, LocPara.Offset, LocPara.KAdc, LocPara.QE, LocPara.ZDepthCorrFactor, LocPara.p4, LocPara.p3, LocPara.p2, LocPara.p1, LocPara.p0, FluoNum);
		break;
	case 17:
		bfgsMLELoc_AS3D<17, FitParaNum_AS3D> << <BlockNum, BlockDim, 0, cstream >> >(d_LocArry, d_SubRegion, d_WLEPara, LocPara.Offset, LocPara.KAdc, LocPara.QE, LocPara.ZDepthCorrFactor, LocPara.p4, LocPara.p3, LocPara.p2, LocPara.p1, LocPara.p0, FluoNum);
		break;


	default:

		break;
	}

	hipStreamQuery(cstream);

}
