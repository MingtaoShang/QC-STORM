/*
This program is free software: you can redistribute it and/or modify
it under the terms of the GNU LESSER GENERAL PUBLIC LICENSE as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU LESSER GENERAL PUBLIC LICENSE for more details.

You should have received a copy of the GNU LESSER GENERAL PUBLIC LICENSE
along with this program.  If not, see <https://www.gnu.org/licenses/>.
*/

#include "cudaWrapper.h"


// cuda wrapper

char AllocHostMemory(void **ptr, long size)
{
	hipError_t err;
	err = hipHostMalloc(ptr, size);

#if bfgs_debug
	HandleErr(err, "malloc host memory");
#endif

	if (err == hipSuccess)return 0;
	else return 1;
}

char FreeHostMemory(void * ptr)
{
	hipError_t err;
	err = hipHostFree(ptr);
#if bfgs_debug
	HandleErr(err, "free host memory");
#endif

	if (err == hipSuccess)return 0;
	else return 1;

}

char AllocGPUMemory(void **ptr, long size)
{
	hipError_t err;
	err = hipMalloc(ptr, size);

#if bfgs_debug
	HandleErr(err, "malloc gpu memory");
#endif

	if (err == hipSuccess)return 0;
	else return 1;
}

char FreeGPUMemory(void * ptr)
{
	hipError_t err;
	err = hipFree(ptr);
#if bfgs_debug
	HandleErr(err, "free gpu memory");
#endif

	if (err == hipSuccess)return 0;
	else return 1;

}

char CreatStream(hipStream_t *pstream)
{
	hipError_t err;
	err = hipStreamCreate(pstream);

#if bfgs_debug
	HandleErr(err, "create stream");
#endif

	if (err == hipSuccess)return 0;
	else return 1;
}

char CreatStreamWithPriority(hipStream_t *pstream, int prio)
{
	hipError_t err;
	err = hipStreamCreateWithPriority(pstream, hipStreamNonBlocking, prio);

#if bfgs_debug
	HandleErr(err, "create stream priority");
#endif

	if (err == hipSuccess)return 0;
	else return 1;
}


char FreeStream(hipStream_t cstream)
{
	hipError_t err;
	err = hipStreamDestroy(cstream);

#if bfgs_debug
	HandleErr(err, "free stream");
#endif

	if (err == hipSuccess)return 0;
	else return 1;
}

// wait current stream to finish the task
char WaitGPUStream(hipStream_t cstream)
{
	hipError_t err;

	err = hipStreamSynchronize(cstream);

#if bfgs_debug
	HandleErr(err, "wait stream");
#endif

	if (err == hipSuccess)return 0;
	else return 1;

}

void HandleErr(hipError_t err, const char * str)
{
	if (err != hipSuccess)
	{
		printf("cuda err:%s, %s\n", str, hipGetErrorString(err));
	}
	else
	{
		printf("cuda suc:%s\n", str);
	}
}

int CudaDeviceNum()
{
	int count;
	hipGetDeviceCount(&count);
	printf("cuda device count:%d\n", count);


	if (count<1 || count>200)
	{
		printf("error: no cuda device\n");
		return 0;
	}

	int cnt;
	for (cnt = 0; cnt<count; cnt++)
	{
		hipDeviceProp_t prop;

		hipGetDeviceProperties(&prop, cnt);

		printf("device name:%s\n", prop.name);

	}
	return count;
}

char CudaSetDevice(int id)
{
	hipError_t err;

	err = hipSetDevice(id);;

#if bfgs_debug
	HandleErr(err, "set device num");
#endif

	if (err == hipSuccess)return 0;
	else return 1;
}
