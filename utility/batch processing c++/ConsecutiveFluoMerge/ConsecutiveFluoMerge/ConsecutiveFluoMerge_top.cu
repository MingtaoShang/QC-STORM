#include "ConsecutiveFluoMerge.h"




void ConsecutiveFluoMerger_TypeDef::MergeConsecutiveFluo(float * h_LocArry, int FluoNum, LocalizationPara & LocPara, int FilterMode, float Distance_th_pixel, hipStream_t cstream)
{

	int BlockDim = ThreadsPerBlock;
	int BlockNum = (FluoNum + ThreadsPerBlock - 1) / ThreadsPerBlock;


	hipMemcpyAsync(d_LocArry, h_LocArry, FluoNum*OutParaNumGS2D*sizeof(float), hipMemcpyHostToDevice, cstream);


	hipMemsetAsync(d_ForwardLinkID, 0, FluoNum * sizeof(int), cstream);
	hipMemsetAsync(d_BackwardLinkID, 0, FluoNum * sizeof(int), cstream);

	
	gpuFindConsecutiveFilterPair << <BlockNum, BlockDim, 0, cstream >> >(d_LocArry, d_ForwardLinkID, d_BackwardLinkID, Distance_th_pixel, FluoNum);

	gpuConsecutiveFit << <BlockNum, BlockDim, 0, cstream >> > (d_LocArry, d_ForwardLinkID, d_BackwardLinkID, LocPara.QE, FluoNum);

	gpuRemoveConsecutiveFluo_KeepFirst << <BlockNum, BlockDim, 0, cstream >> >(d_LocArry, d_ForwardLinkID, d_BackwardLinkID, FluoNum);

	// localization precision calculated by CRLB
	LDLocData_TypeDef::LocPrecCalc_GaussianCRLB(d_LocArry, LocPara, FluoNum, cstream);

	hipMemcpyAsync(h_LocArry, d_LocArry, FluoNum*OutParaNumGS2D*sizeof(float), hipMemcpyDeviceToHost, cstream);


	hipStreamSynchronize(cstream);

}



void ConsecutiveFluoMerger_TypeDef::Init(unsigned int TotalFluoNum)
{
	hipError_t err;

	printf("TotalFluoNum:%d\n", TotalFluoNum);

	err = hipHostMalloc((void **)&h_LocArry, TotalFluoNum * OutParaNumGS2D * sizeof(float));
	HandleErr(err,"hipHostMalloc h_LocArry");

	err = hipMalloc((void **)&d_LocArry, TotalFluoNum * OutParaNumGS2D*sizeof(float));
	HandleErr(err, "hipMalloc h_LocArry");


	hipMalloc((void **)&d_ForwardLinkID, TotalFluoNum * sizeof(int));
	hipMalloc((void **)&d_BackwardLinkID, TotalFluoNum * sizeof(int));

}


void ConsecutiveFluoMerger_TypeDef::DeInit()
{
	hipHostFree(h_LocArry);
	hipFree(d_LocArry);


	hipFree(d_ForwardLinkID);
	hipFree(d_BackwardLinkID);

}


